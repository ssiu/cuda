
#include <hipcub/hipcub.hpp>


void sum_cub(float* d_in, float* d_out, int N) {

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, N);
//    hipMalloc(&d_temp_storage, temp_storage_bytes);
//    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, N);
//    hipFree(d_temp_storage);

}