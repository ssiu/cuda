#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cute/tensor.hpp>
#include <cute/algorithm/gemm.hpp>
#include <cute/arch/mma.hpp>
// using cute machinery to for 1x1x1


// initial a 1x1x1 matrices


// copy 1x1x1 matrix
// copy traits
using namespace cute;

__global__ void mma(float* A, float* B, float* C) {
    printf("A = %f, B = %f\n", A[0], B[0]);
    //gemm(C[0], A[0], B[0], C[0]);

}
// do mma
// mma traits


int main() {

    // Allocate memory on the host
    thrust::host_vector<float> h_A(1);
    thrust::host_vector<float> h_B(1);
    thrust::host_vector<float> h_C(1);

    // Initialize matrices h_A and h_B with data
    h_A[0] = 2.0f;
    h_B[0] = 3.0f;
    h_C[0] = 0.0f;

    thrust::device_vector<float> d_A = h_A;
    thrust::device_vector<float> d_B = h_B;
    thrust::device_vector<float> d_C = h_C;

    //call mma
    mma<<<1,1>>>(d_A.data().get(), d_B.data().get(), d_C.data().get());

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        //goto Error; // Use appropriate error handling here
    }


    h_C = d_C;
    printf("C = %f \n", h_C[0]);



    return 0;
}