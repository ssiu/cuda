#include "hip/hip_runtime.h"
// using retile_D

#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cute/tensor.hpp>
#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"
#include "utils.cuh"

using namespace cute;

template <class ASmemLayout, class TiledCopyA,
          class BSmemLayout, class TiledCopyB,
          class CSmemLayout, class TiledMma>
__global__ void mm_kernel(
           half_t* A, ASmemLayout sA_layout, TiledCopyA copy_a,
           half_t* B, BSmemLayout sB_layout, TiledCopyB copy_b,
           float*  C, CSmemLayout sC_layout, TiledMma      mma)
{

    Tensor gA = make_tensor(make_gmem_ptr(A), sA_layout);
    Tensor gB = make_tensor(make_gmem_ptr(B), sB_layout);
    Tensor gC = make_tensor(make_gmem_ptr(C), sC_layout);

    __shared__ half_t smemA[cosize_v<ASmemLayout>];
    __shared__ half_t smemB[cosize_v<BSmemLayout>];

    Tensor sA = make_tensor(make_smem_ptr(smemA), sA_layout);
    Tensor sB = make_tensor(make_smem_ptr(smemB), sB_layout);

    ThrCopy thr_copy_a = copy_a.get_slice(threadIdx.x);
    Tensor tAgA = thr_copy_a.partition_S(gA);                            // (CPY,CPY_M,CPY_K,k)
    Tensor tAsA = thr_copy_a.partition_D(sA);                            // (CPY,CPY_M,CPY_K)

    ThrCopy thr_copy_b = copy_b.get_slice(threadIdx.x);
    Tensor tBgB = thr_copy_b.partition_S(gB);                            // (CPY,CPY_N,CPY_K,k)
    Tensor tBsB = thr_copy_b.partition_D(sB);                            // (CPY,CPY_N,CPY_K)


    ThrMMA thr_mma = mma.get_slice(threadIdx.x);
    Tensor tCrA = thr_mma.partition_fragment_A(sA);
    Tensor tCrB = thr_mma.partition_fragment_B(sB);

    auto smem_tiled_copy_A = make_tiled_copy_A(Copy_Atom<DefaultCopy, half_t>{}, tiled_mma);
    auto smem_thr_copy_A   = smem_tiled_copy_A.get_thread_slice(thread_idx);
    Tensor tCsA            = smem_thr_copy_A.partition_S(sA);                  // (CPY,CPY_M,CPY_K,PIPE)
    Tensor tCrA_copy_view  = smem_thr_copy_A.retile_D(tCrA);                   // (CPY,CPY_M,CPY_K)


    auto smem_tiled_copy_B = make_tiled_copy_B(Copy_Atom<DefaultCopy, half_t>{}, tiled_mma);
    auto smem_thr_copy_B   = smem_tiled_copy_B.get_thread_slice(thread_idx);
    Tensor tCsB            = smem_thr_copy_B.partition_S(sB);                  // (CPY,CPY_N,CPY_K,PIPE)
    Tensor tCrB_copy_view  = smem_thr_copy_B.retile_D(tCrB);                   // (CPY,CPY_N,CPY_K)





//     Tensor tCsA = thr_mma.partition_A(sA);                               // (MMA,MMA_M,MMA_K)
//     Tensor tCsB = thr_mma.partition_B(sB);                               // (MMA,MMA_N,MMA_K)
//     Tensor tCgC = thr_mma.partition_C(gC);                               // (MMA,MMA_M,MMA_N)

    // Allocate the accumulators -- same size as the projected data
    Tensor tCrC = thr_mma.make_fragment_C(tCgC);

    //printf("tCrC: %f\n", tCrC[0]);
    clear(tCrC);

    copy(copy_a, tAgA, tAsA);
    copy(copy_b, tBgB, tBsB);

    __syncthreads();

    copy(smem_tiled_copy_A, tCsA, tCrA_copy_view);
    copy(smem_tiled_copy_B, tCsB, tCrB_copy_view);

    gemm(mma, tCrA, tCrB, tCrC);

    axpby(1.0f, tCrC, 0.0f, tCgC); //test

    #if 0
        if(thread0()) {
        print("  gA : "); print(  gA); print("\n");
        print("  sA : "); print(  sA); print("\n");
        print("tAgA : "); print(tAgA); print("\n");
        print("tAsA : "); print(tAsA); print("\n");

        }
    #endif

    #if 0
        if(thread0()) {
        print("  gB : "); print(  gB); print("\n");
        print("  sB : "); print(  sB); print("\n");
        print("tBgB : "); print(tBgB); print("\n");
        print("tBsB : "); print(tBsB); print("\n");
        }
    #endif

    #if 0
        if(thread(0)) {
            print("  gC : "); print(  gC); print("\n");
            print("tCsA : "); print(tCsA); print("\n");
            print("tCsB : "); print(tCsB); print("\n");
            print("tCgC : "); print(tCgC); print("\n");
            print("tCrC : "); print(tCrC); print("\n");
            printf("tCsA[0], sA[0]: %f %f\n", static_cast<float>(tCsA[0]),static_cast<float>(sA[0]));
            printf("tCsA[1], sA[16]: %f %f\n", static_cast<float>(tCsA[1]),static_cast<float>(sA[16]));
            printf("tCsA[2], sA[8]: %f %f\n", static_cast<float>(tCsA[2]),static_cast<float>(sA[8]));
            printf("tCsA[3], sA[24]: %f %f\n", static_cast<float>(tCsA[3]),static_cast<float>(sA[24]));
            printf("tCsB[0], sB[0]: %f %f\n", static_cast<float>(tCsB[0]),static_cast<float>(sB[0]));
            printf("tCsB[1], sB[1]: %f %f\n", static_cast<float>(tCsB[1]),static_cast<float>(sB[1]));
//             for (int i=0;i< 16; i++) {
//                 for (int j=0;j<8;j++) {
//                     printf("%f ", static_cast<float>(sA[i  + 16 * j]));
//                 }
//                 printf("\n");
//             }

        }
    #endif

    #if 0
        printf("thread = %d, tCsB[0] = %f\n", threadIdx.x, static_cast<float>(tCsB[0]));
        printf("thread = %d, tCsB[1] = %f\n", threadIdx.x, static_cast<float>(tCsB[1]));
    #endif

    #if 0
        if(thread0()) {
            for (int i=0; i<128; i++){
                printf("i = %d, gA = %f, sA = %f,\n", i, static_cast<float>(gA[i]), static_cast<float>(sA[i]));
            }
            for (int i=0; i<64; i++){
                printf("i = %d, gB = %f, sB = %f,\n", i, static_cast<float>(gB[i]), static_cast<float>(sB[i]));
            }
        }
    #endif

}


void mm(half_t* A, half_t* B, float* C) {

    auto sA_layout = make_layout(make_shape (Int<128>{}, Int<8>{}),
                        make_stride(Int<1>{}, Int<128>{}));
//     auto sB_layout = make_layout(make_shape (Int<8>{}, Int<8>{}),
//                         make_stride(Int<1>{}, Int<8>{}));
    auto sB_layout = make_layout(make_shape (Int<128>{}, Int<8>{}),
                        make_stride(Int<8>{}, Int<1>{}));
    auto sC_layout = make_layout(make_shape (Int<128>{}, Int<128>{}),
                        make_stride(Int<1>{}, Int<128>{}));

    TiledCopy copyA = make_tiled_copy(Copy_Atom<DefaultCopy, half_t>{},
                               Layout<Shape<_32,_8>, Stride<_1,_32>>{},
                               Layout<Shape< _4,_1>>{});
    TiledCopy copyB = make_tiled_copy(Copy_Atom<DefaultCopy, half_t>{},
                               Layout<Shape<_128,_2>, Stride<_2,_1>>{},
                               Layout<Shape< _1,_4>>{});
    TiledMMA mmaC = make_tiled_mma(SM75_16x8x8_F32F16F16F32_TN{},
                                    Layout<Shape<_2, _4, _1>>{},
                                    Tile<_128,_128,_8>{});

    dim3 dimGrid(1);
    dim3 dimBlock(256);
    mm_kernel<<<dimGrid, dimBlock>>>(A, sA_layout, copyA,
                                     B, sB_layout, copyB,
                                     C, sC_layout, mmaC);
}



void mm_cublas(half_t* A, half_t* B, float* C,
                int M, int N, int K) {
    float alpha = 1.0f;
    float beta = 0.0f;

    hipError_t cudaStat;  // hipMalloc status
    hipblasStatus_t stat;   // cuBLAS functions status
    hipblasHandle_t handle; // cuBLAS context
    hipblasCreate(&handle);
    //hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 16, 8, 8, &alpha, A, 16, B, 8, &beta, C, 16);
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
                            A, HIP_R_16F, M,
                            B, HIP_R_16F, K, &beta,
                            C, HIP_R_32F, M,
                            HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
    hipblasDestroy(handle);
}


void mm_cpu(float* A, float* B, float* C,
            int M, int N, int K) {
    for (int k = 0; k < K; k ++) {
        for (int i=0; i< M; i++) {
            for (int j=0; j < N; j++) {
                C[i + M * j] += A[i + M * k] * B[k + K * j];
            }
        }
    }
}

int main(int argc, char** argv)
{
    int m = 128;
    int n = 128;
    int k = 8;

    using TA = half_t;
    using TB = half_t;
    using TC = float;

    cute::device_init(0);

    thrust::host_vector<TA> h_A(m*k);
    thrust::host_vector<TB> h_B(n*k);
    thrust::host_vector<TC> h_C(m*n);
    thrust::host_vector<TC> h_C_cublas(m*n);

    thrust::host_vector<TC> h_A_cpu(m*k);
    thrust::host_vector<TC> h_B_cpu(n*k);
    thrust::host_vector<TC> h_C_cpu(m*n);

    for (int j = 0; j < m*k; ++j) {
        //h_A[j] = static_cast<TA>(j);
        h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
//         if (j==16) {
//             h_A[j] = static_cast<TA>(1);
//         } else {
//             h_A[j] = static_cast<TA>(0);
//         }
        h_A_cpu[j] = static_cast<float>(h_A[j]);
    }
    for (int j = 0; j < n*k; ++j) {
        //h_B[j] = static_cast<TB>(j);
//         if (j==1) {
//             h_B[j] = static_cast<TB>(1);
//         } else {
//             h_B[j] = static_cast<TB>(0);
//         }
        h_B[j] = static_cast<TB>( 2*(rand() / double(RAND_MAX)) - 1 );
        h_B_cpu[j] = static_cast<float>(h_B[j]);
    }
    for (int j = 0; j < m*n; ++j) {
        h_C[j] = static_cast<TC>(0);
        h_C_cublas[j] = static_cast<TC>(0);
        h_C_cpu[j] = static_cast<TC>(0);
    }

    thrust::device_vector<TA> d_A = h_A;
    thrust::device_vector<TB> d_B = h_B;
    thrust::device_vector<TC> d_C = h_C;
    thrust::device_vector<TC> d_C_cublas = h_C_cublas;

    mm(d_A.data().get(), d_B.data().get(), d_C.data().get());
    mm_cublas(d_A.data().get(), d_B.data().get(), d_C_cublas.data().get(), m, n, k);
    mm_cpu(h_A_cpu.data(), h_B_cpu.data(), h_C_cpu.data(), m, n, k);
//
//     thrust::host_vector<TC> h_C_result = d_C;
//     thrust::host_vector<TC> h_C_cublas_result = d_C_cublas;
    h_C = d_C;
    h_C_cublas = d_C_cublas;


    if (isSameMatrices(h_C.data(), h_C_cpu.data(), m, n) && isSameMatrices(h_C.data(), h_C_cublas.data(), m, n)) {
        printf("Correct answer\n");
    } else {
        printf("Wrong answer\n");
    }

    #if 0
        for (int i=0;i< 16; i++) {
            for (int j=0;j<8;j++) {
                printf("%f ", h_C_result[i + 16 * j]);
            }
            printf("\n");
        }
    #endif

    #if 0
        for (int i=0; i<128; i++){
        //for (int i=0; i<32; i++){
            printf("i = %d, cutlass = %f, cublas = %f, cpu = %f\n", i, h_C_result[i], h_C_cublas_result[i], h_C_cpu[i]);
        }
    #endif


    return 0;
}
