#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cute/tensor.hpp>
#include <cute/algorithm/gemm.hpp>
#include <cute/algorithm/copy.hpp>
#include <cute/arch/mma.hpp>

using namespace cute;


// partition_fragment_A and partition_fragment_B often depend on the
//   layout of A and B and/or the thread_idx that is requesting the partition.
// For these reasons, they should not be used in a static context.
// See TiledMMA::get_slice(thr_idx).partition_fragment_A(tensorA) instead.
__global__ void mma_atom(float* dA, float* dB, float* dC) {
    //printf("A = %f, B = %f\n", dA[0], dB[0]);
    //gemm(C[0], A[0], B[0], C[0]);

    auto gA = make_tensor(make_gmem_ptr(dA), make_shape(Int<2>{}, Int<1>{}), make_stride(Int<1>{}, Int<1>{}));      // (M,K)
    auto gB = make_tensor(make_gmem_ptr(dB), make_shape(Int<2>{}, Int<1>{}), make_stride(Int<1>{}, Int<1>{}));      // (N,K)
    auto gC = make_tensor(make_gmem_ptr(dC), make_shape(Int<2>{}, Int<2>{}), make_stride(Int<1>{}, Int<2>{}));      // (M,N)

    using Mma_atom = MMA_Atom<MMA_Traits<UniversalFMA<float,float,float,float>>>;

    using TiledMma = TiledMMA<
      Mma_atom,
      Layout<Shape<_2,_2,_1>>,  // 2x2x1 thread group
      Layout<Shape<_1,_1,_1>>>; // 1x2x1 value group for 16x16x16 MMA and LDSM

    TiledMma tiled_mma;

    auto thr_mma = tiled_mma.get_thread_slice(threadIdx.x);
    Tensor rA  = thr_mma.partition_fragment_A(gA);
    Tensor rB  = thr_mma.partition_fragment_B(gB);
    Tensor rC  = thr_mma.partition_fragment_C(gC);

    copy(gA, rA);
    copy(gB, rB);

    if {threadIdx.x == 0} {
        print_tensor(rA);
        print_tensor(rB);
        print_tensor(rC);
    }


//    print_tensor(gA);
//    auto rA = make_fragment_like(gA);
//    auto rB = make_fragment_like(gB);
//    auto rC = make_fragment_like(gC);
//
//
//    copy(gA, rA);
//    copy(gB, rB);
//
//    print_tensor(rA);
//
//    gemm(mma, rA, rB, rC);
//    copy(rC, gC);

//    printf("rA = %f, rB = %f, rC = %f\n", rA[0], rB[0], rC[0]);

}


int main() {

    // Allocate memory on the host
    thrust::host_vector<float> hA(2);
    thrust::host_vector<float> hB(2);
    thrust::host_vector<float> hC(4);

    // Initialize matrices h_A and h_B with data
    hA[0] = 1.0f;
    hA[1] = 2.0f;
    hB[0] = 3.0f;
    hB[1] = 5.0f;
    hC[0] = 0.0f;
    hC[1] = 0.0f;
    hC[2] = 0.0f;
    hC[3] = 0.0f;

    thrust::device_vector<float> dA = hA;
    thrust::device_vector<float> dB = hB;
    thrust::device_vector<float> dC = hC;

    //call mma
    mma_atom<<<1,4>>>(dA.data().get(), dB.data().get(), dC.data().get());

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        //goto Error; // Use appropriate error handling here
    }


    hC = dC;
    printf("C = %f \n", hC[0]);



    return 0;
}