// naive matrix multiplication

#include <hip/hip_runtime.h>
#include <iostream>


__global__ void mm_global_memory_coalescing_kernel(float* A, float* B, float* C, int N){
    //int threadId = threadIdx.x + blockDim.x * threadIdx.y
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    float sum = 0;
    for (int i = 0; i < N; i++){
        sum += A[row*N + i] * B[i*N + col];
//        if (blockIdx.x == 0 and blockIdx.x==0 and threadIdx.x==0 and threadIdx.y==0){
//            printf("A is %f, B is %f, SUM is %f\n", A[row*N + i], B[i*N + col], sum);
//        }
    }
    C[row*N + col] = sum;
}

void mm_global_memory_coalescing(float* A, float* B, float* C, int N) {
    dim3 dimGrid(N / 32, N / 32);
    dim3 dimBlock(32, 32);
    mm_global_memory_coalescing_kernel<<<dimGrid, dimBlock>>>(A, B, C, N);
}