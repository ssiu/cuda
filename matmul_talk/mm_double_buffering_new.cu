
#include <hip/hip_runtime.h>
#include <iostream>
#define A(i,j) A[(i) * N + (j)]
#define B(i,j) B[(i) * N + (j)]
#define C(i,j) C[(i) * N + (j)]
#define sA(pointer, i,j) sA[(pointer)][((i) << 7) + (j)]
#define sB(pointer, i,j) sB[(pointer)][((i) << 7) + (j)]
#define TILE_WIDTH 128
#define BLOCK_WIDTH 8
#define FLOAT_4(pointer) reinterpret_cast<float4*>(&(pointer))[0]


__global__ __launch_bounds__(256,2)
void mm_double_buffering_new_kernel(float* A, float* B, float* C, int N){
    int thread_id = threadIdx.x;
    int block_idx = blockIdx.x;
    int block_idy = blockIdx.y;
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 31;
    int warp_row = (warp_id >> 1) << 5;
    int warp_col = (warp_id & 1) << 6;
    int thread_row = (lane_id >> 3) << 2;
    int thread_col = (lane_id & 7) << 2;


    int sA_row = thread_id >> 1;
    int sA_col = (thread_id & 1) << 2;

    int sB_row = thread_id >> 5;
    int sB_col = (thread_id & 31) << 2;


    int permuted_warp_id = (warp_id ) ^ (thread_id & 1);
    int permuted_thread_id = (permuted_warp_id << 5) + lane_id;
    int permuted_sA_row = permuted_thread_id >> 1;

    int C_row = warp_row + thread_row;
    int C_col = warp_col + thread_col;


    A = &A((block_idx << 7), 0);
    B = &B(0, (block_idy << 7));
    C = &C((block_idx << 7), (block_idy << 7));

    __shared__ float sA[2][BLOCK_WIDTH * TILE_WIDTH];
    __shared__ float sB[2][BLOCK_WIDTH * TILE_WIDTH];


    float rA[4];
    float rB[4];

    float fA[8] = {};
    float fB[8] = {};

    float accum[64] = {};

    int shared_pointer = 0;
    // load first block
    FLOAT_4(rA) = FLOAT_4(A(sA_row, sA_col));
    FLOAT_4(rB) = FLOAT_4(B(sB_row, sB_col));
    #pragma unroll
    for (int i=0; i<4;i++){
        sA(shared_pointer, sA_col + i, permuted_sA_row) = rA[i];
    }

    FLOAT_4(sB(shared_pointer, sB_row, sB_col)) = FLOAT_4(rB);

    __syncthreads();

    A += BLOCK_WIDTH;
    B += BLOCK_WIDTH * N;

    for (int kBlock=0; kBlock<N/BLOCK_WIDTH; kBlock++){

        // load from gmem A, B for next block
        if (kBlock < N/BLOCK_WIDTH - 1) {
            FLOAT_4(rA) = FLOAT_4(A(sA_row, sA_col));
            FLOAT_4(rB) = FLOAT_4(B(sB_row, sB_col));
        }
        #pragma unroll
        for (int kFragment=0; kFragment<4; kFragment++) {
            // load from smem A, B
            FLOAT_4(fA[0]) = FLOAT_4(sA(shared_pointer, kFragment, C_row));
            FLOAT_4(fA[4]) = FLOAT_4(sA(shared_pointer, kFragment, C_row + 16));
            FLOAT_4(fB[0]) = FLOAT_4(sB(shared_pointer, kFragment, C_col));
            FLOAT_4(fB[4]) = FLOAT_4(sB(shared_pointer, kFragment, C_col + 32));
            // compute outer product
            #pragma unroll
            for (int i=0; i<8;i++){
                #pragma unroll
                for (int j=0; j<8; j++) {
                    accum[i*8+j] += fA[i] * fB[j];
                }
             }

        }

        #pragma unroll
        for (int kFragment=4; kFragment<BLOCK_WIDTH; kFragment++) {
            // load from smem A, B
            FLOAT_4(fA[0]) = FLOAT_4(sA(shared_pointer, kFragment, C_row + 16));
            FLOAT_4(fA[4]) = FLOAT_4(sA(shared_pointer, kFragment, C_row));
            FLOAT_4(fB[0]) = FLOAT_4(sB(shared_pointer, kFragment, C_col));
            FLOAT_4(fB[4]) = FLOAT_4(sB(shared_pointer, kFragment, C_col + 32));
            // compute outer product
            #pragma unroll
            for (int i=0; i<8;i++){
                #pragma unroll
                for (int j=0; j<8; j++) {
                    accum[i*8+j] += fA[i] * fB[j];
                }
             }

        }
        // store to smem sA, sB for next block
        if (kBlock < N/BLOCK_WIDTH - 1) {


            //FLOAT_4(sA[sA_sOffset]) = FLOAT_4(rA);
            #pragma unroll
            for (int i=0; i<4;i++){
                sA(shared_pointer^1, sA_col + i, permuted_sA_row) = rA[i];
                //sA[shared_pointer^1][sA_sOffset + i*TILE_WIDTH] = rA[i];
            }

            FLOAT_4(sB(shared_pointer^1, sB_row, sB_col)) = FLOAT_4(rB);

            __syncthreads();

            A += BLOCK_WIDTH;
            B += BLOCK_WIDTH * N;

            shared_pointer ^= 1;
        }

    }

//    storeToGmem_5(accum, C, N, C_gOffset);

    // store to gmem C
    #pragma unroll
    for (int i=0;i<4;i++) {

        FLOAT_4(C(C_row + i, C_col)) = FLOAT_4(accum[i * 8]);
        FLOAT_4(C(C_row + i, C_col + 32)) = FLOAT_4(accum[i * 8 + 4]);
        FLOAT_4(C(C_row + i + 16, C_col)) = FLOAT_4(accum[(i+4) * 8]);
        FLOAT_4(C(C_row + i + 16, C_col + 32)) = FLOAT_4(accum[(i+4) * 8 + 4]);

    }
}


void mm_double_buffering_new(float* A, float* B, float* C, int N) {
    dim3 dimGrid(N / TILE_WIDTH, N / TILE_WIDTH);
    dim3 dimBlock(256);
    mm_double_buffering_new_kernel<<<dimGrid, dimBlock>>>(A, B, C, N);
}