// matrix multiplication
// global memory coalescing
// shared memory blocking

#include <hip/hip_runtime.h>
#include <iostream>

#define TILE_WIDTH 32
__global__ void mm_shared_memory_tiling_kernel(float* A, float* B, float* C, int N){
    //int threadId = threadIdx.x + blockDim.x * threadIdx.y
    // rows and columns that the thread compute the in output C matrix
    int cRow = threadIdx.y + blockDim.y * blockIdx.y;
    int cCol = threadIdx.x + blockDim.x * blockIdx.x;
    // rows and columns in shared memory;
    int sRow = threadIdx.y;
    int sCol = threadIdx.x;
    int gRow_A = cRow;
    int gCol_A;
    int gRow_B;
    int gCol_B = cCol;

    __shared__ float sA[TILE_WIDTH*TILE_WIDTH];
    __shared__ float sB[TILE_WIDTH*TILE_WIDTH];

    // load into shared memory
    float sum = 0;
    for (int kTile=0; kTile < N/TILE_WIDTH; kTile++){
        //offset is row, kTile*TILE_WIDTH
        gCol_A = kTile*TILE_WIDTH + threadIdx.x;
        gRow_B = kTile*TILE_WIDTH + threadIdx.y;
        sA[sRow * TILE_WIDTH + sCol] = A[gRow_A * N + gCol_A];
        sB[sRow * TILE_WIDTH + sCol] = B[gRow_B * N + gCol_B];
        __syncthreads();


        for (int i=0; i<TILE_WIDTH; i++){
//            if (blockIdx.x == 0 and blockIdx.x==0 and threadIdx.x==0 and threadIdx.y==0){
//                printf("SUM is %f\n", sum);
//            }
            sum += sA[sRow*TILE_WIDTH + i] * sB[i*TILE_WIDTH + sCol];
        }
        __syncthreads();
    }

    C[cRow*N + cCol] = sum;
}

void mm_shared_memory_tiling(float* A, float* B, float* C, int N) {
    dim3 dimGrid(N / 32, N / 32);
    dim3 dimBlock(32, 32);
    mm_shared_memory_tiling_kernel<<<dimGrid, dimBlock>>>(A, B, C, N);
}