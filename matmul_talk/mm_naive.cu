// naive matrix multiplication

#include <hip/hip_runtime.h>
#include <iostream>


__global__ void mm_naive_kernel(float* A, float* B, float* C, int N){
    //int threadId = threadIdx.x + blockDim.x * threadIdx.y
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    float sum = 0;
    for (int i = 0; i < N; i++){
        sum += A[row*N + i] * B[i*N + col];
//        if (blockIdx.x == 0 and blockIdx.x==0 and threadIdx.x==0 and threadIdx.y==0){
//            printf("A is %f, B is %f, SUM is %f\n", A[row*N + i], B[i*N + col], sum);
//        }
    }
    C[row*N + col] = sum;
}

void mm_naive(float* A, float* B, float* C, int N) {
    dim3 dimGrid(N / 32, N / 32);
    dim3 dimBlock(32, 32);
    mm_naive_kernel<<<dimGrid, dimBlock>>>(A, B, C, N);
}