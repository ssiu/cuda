
#include <hip/hip_runtime.h>
#include <hipblas.h>

void mm_cublas(float* A, float* B, float* C, int N) {
    float alpha = 1.0f;
    float beta = 1.0f;

    hipError_t cudaStat;  // cudaMalloc status
    hipblasStatus_t stat;   // cuBLAS functions status
    hipblasHandle_t handle; // cuBLAS context
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, B, N, A, N, &beta, C, N);

    hipblasDestroy(handle);
}

