#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "utils.cuh"
#include "sm75_gemm_test.cu"
#include "sm75_gemm_vectorized_load.cu"
#include "sm75_gemm_swizzle.cu"
#include "sm75_gemm_ldsm.cu"
#include "sm75_gemm_smem_buffering.cu"



#include "sm75_gemm_vectorized_load_256.cu"
#include "sm75_gemm_vectorized_gmem_store_256.cu"
#include "sm75_gemm_swizzle_256.cu"
#include "sm75_gemm_ldsm_256.cu"
#include "sm75_gemm_smem_pipelining_256.cu"
#include "sm75_gemm_register_pipelining_256.cu"



using namespace cute;


int main(int argc, char** argv)
{
    int m = 128;
    if (argc >= 2)
    sscanf(argv[1], "%d", &m);

    int n = 128;
    if (argc >= 3)
    sscanf(argv[2], "%d", &n);

    int k = 32;
    if (argc >= 4)
    sscanf(argv[3], "%d", &k);

    using TA = half_t;
    using TB = half_t;
    using TC = float;

    cute::device_init(0);

    thrust::host_vector<TA> h_A = generateRandomMatrix<TA> (m * k);
    thrust::host_vector<TB> h_B = generateRandomMatrix<TB> (n * k);
    thrust::host_vector<TC> h_C(m * n, 0.0f);
    thrust::host_vector<TC> h_C_cublas(m * n, 0.0f);


    thrust::device_vector<TA> d_A = h_A;
    thrust::device_vector<TB> d_B = h_B;
    thrust::device_vector<TC> d_C = h_C;
    thrust::device_vector<TC> d_C_cublas = h_C_cublas;

    //gemm_test(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);


    gemm_cublas(d_A.data().get(), d_B.data().get(), d_C_cublas.data().get(), m, n, k);
    h_C_cublas = d_C_cublas;
//
//
    //if (isSameMatrices(h_C.data(), h_C_cpu.data(), m, n) && isSameMatrices(h_C.data(), h_C_cublas.data(), m, n)) {
//     gemm_vectorized_load(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);
//     h_C = d_C;
//     isSameMatrices(h_C.data(), h_C_cublas.data(), m * n, "vectorized_load");
//
//     gemm_swizzle(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);
//     h_C = d_C;
//     isSameMatrices(h_C.data(), h_C_cublas.data(), m * n, "swizzle");
//
//     gemm_ldsm(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);
//     h_C = d_C;
//     isSameMatrices(h_C.data(), h_C_cublas.data(), m * n, "ldsm");
// //
//     gemm_smem_buffering(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);
//     h_C = d_C;
//     isSameMatrices(h_C.data(), h_C_cublas.data(), m * n, "smem_buffering");
//
//     gemm_test(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);
//     h_C = d_C;
//     isSameMatrices(h_C.data(), h_C_cublas.data(), m * n, "test");
//
    gemm_vectorized_load_256(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);
    h_C = d_C;
    isSameMatrices(h_C.data(), h_C_cublas.data(), m * n, "vectorized_load_256");

    gemm_vectorized_gmem_store_256(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);
    h_C = d_C;
    isSameMatrices(h_C.data(), h_C_cublas.data(), m * n, "vectorized_gmem_store_256");

//     for (int i=0;i<32;i++) {
//         printf("cutlass = %f, cublas = %f\n", h_C[i], h_C_cublas[i]);
//     }

    gemm_swizzle_256(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);
    h_C = d_C;
    isSameMatrices(h_C.data(), h_C_cublas.data(), m * n, "swizzle_256");

//     for (int i=0;i<32;i++) {
//         printf("cutlass = %f, cublas = %f\n", h_C[i], h_C_cublas[i]);
//     }

    gemm_ldsm_256(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);
    h_C = d_C;
    isSameMatrices(h_C.data(), h_C_cublas.data(), m * n, "ldsm_256");

    gemm_smem_pipelining_256(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);
    h_C = d_C;
    isSameMatrices(h_C.data(), h_C_cublas.data(), m * n, "smem_pipelining_256");

    gemm_register_pipelining_256(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, k);
    h_C = d_C;
    isSameMatrices(h_C.data(), h_C_cublas.data(), m * n, "smem_register_256");



    return 0;
}