#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <random>

__global__ void perform_float_operations(float* a, float* b, float* c) {
    float ra = a[0];
    float rb = b[0];
    c[0] = ra + rb;
    c[1] = ra * rb;
    c[2] = ra / rb;
}

__global__ void perform_int_operations(int* a, int* b, int* c) {
    int ra = a[0];
    int rb = b[0];
    c[0] = ra + rb;
    c[1] = ra * rb;
    c[2] = ra / rb;
    c[3] = ra % rb;
    c[4] = ra >> 2;
    c[5] = ra & 3;
}


__global__ void int_add(int* a, int* b, int* c) {
    c[0] = a[0] + b[0];
}

__global__ void int_mul(int* a, int* b, int* c) {
    c[0] = a[0] * b[0];
}

__global__ void int_div(int* a, int* b, int* c) {
    c[0] = a[0] / b[0];
}

__global__ void int_mod(int* a, int* b, int* c) {
    c[0] = a[0] % b[0];
}

__global__ void int_shf(int* a, int* c) {
    c[0] = a[0] >> 2;
}

__global__ void int_and(int* a, int* c) {
    c[0] = a[0] & 3;
}





int main() {
    float fa = 8.0f;
    float fb = 3.0f;
    int ia = 8;
    int ib = 4;
    thrust::host_vector<float> a(1, fa);
    thrust::host_vector<float> b(1, fb);
//    thrust::host_vector<float> a(1, fa);
//    thrust::host_vector<float> b(1, fb);
    thrust::host_vector<float> c(3);
    thrust::host_vector<int> a_int(1, ia);
    thrust::host_vector<int> b_int(1, ib);
    thrust::host_vector<int> c_int(6);


    thrust::device_vector<float> da = a;
    thrust::device_vector<float> db = b;
    thrust::device_vector<float> dc = c;
    thrust::device_vector<int> da_int = a_int;
    thrust::device_vector<int> db_int = b_int;
    thrust::device_vector<int> dc_int = c_int;


    dim3 blockDim(1);
    dim3 gridDim(1);

    perform_float_operations<<<gridDim, blockDim>>>(thrust::raw_pointer_cast(da.data()), thrust::raw_pointer_cast(db.data()), thrust::raw_pointer_cast(dc.data()));
    perform_int_operations<<<gridDim, blockDim>>>(thrust::raw_pointer_cast(da_int.data()), thrust::raw_pointer_cast(db_int.data()), thrust::raw_pointer_cast(dc_int.data()));
    int_add<<<gridDim, blockDim>>>(thrust::raw_pointer_cast(da_int.data()), thrust::raw_pointer_cast(db_int.data()), thrust::raw_pointer_cast(dc_int.data()));
    int_mul<<<gridDim, blockDim>>>(thrust::raw_pointer_cast(da_int.data()), thrust::raw_pointer_cast(db_int.data()), thrust::raw_pointer_cast(dc_int.data()));
    int_div<<<gridDim, blockDim>>>(thrust::raw_pointer_cast(da_int.data()), thrust::raw_pointer_cast(db_int.data()), thrust::raw_pointer_cast(dc_int.data()));
    int_mod<<<gridDim, blockDim>>>(thrust::raw_pointer_cast(da_int.data()), thrust::raw_pointer_cast(db_int.data()), thrust::raw_pointer_cast(dc_int.data()));
    int_shf<<<gridDim, blockDim>>>(thrust::raw_pointer_cast(da_int.data()), thrust::raw_pointer_cast(dc_int.data()));
    int_and<<<gridDim, blockDim>>>(thrust::raw_pointer_cast(da_int.data()), thrust::raw_pointer_cast(dc_int.data()));

    c = dc;
    c_int = dc_int;
    std::cout << "float" << std::endl;
    for (int i=0;i<6;i++){
        std::cout << c[i] << std::endl;
    }
    std::cout << "int" << std::endl;
    for (int i=0;i<6;i++){
        std::cout << c_int[i] << std::endl;
    }
    return 0;

}