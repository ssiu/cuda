
#include <hip/hip_runtime.h>
#include <iostream>
// 1024 threads computing a 128*128 block
#define TILE_WIDTH 128
#define BLOCK_WIDTH 8


__device__ void loadFromGmem_3(float* gM, float* r, int offset){
    reinterpret_cast<float4*>(r)[0] = reinterpret_cast<float4*>(&gM[offset])[0];
}

__device__ void storeToSmem_3(float* r, float* sM, int offset){
    reinterpret_cast<float4*>(&sM[offset])[0] = reinterpret_cast<float4*>(r)[0];
}

__device__ void loadFromSmemA_3(float* sM, float* f, int offset){
    for (int i=0; i<4; i++) {
        f[i] = sM[offset + i * BLOCK_WIDTH];
        f[i+4] = sM[offset + (i + 16) * BLOCK_WIDTH];
    }
}

__device__ void loadFromSmemB_3(float* sM, float* f, int offset){
    for (int i=0; i<4; i++) {
        f[i] = sM[offset + i];
        f[i+4] = sM[offset + i + 32];
    }

}

__device__ void computeOuterProduct_3(float* fA, float* fB, float* accum){
    for (int i=0; i<8;i++){
        for (int j=0; j<8; j++) {
            accum[i*8+j] += fA[i] * fB[j];
        }
    }
}

__device__ void storeToGmem_3(float* accum, float* C, int N, int offset){
    for (int i=0;i<4;i++) {
        reinterpret_cast<float4*>(&C[offset + i * N])[0] = reinterpret_cast<float4*>(&accum[i * 8])[0];
        reinterpret_cast<float4*>(&C[offset + i * N + 32])[0] = reinterpret_cast<float4*>(&accum[i * 8 + 4])[0];
        reinterpret_cast<float4*>(&C[offset + (i + 16) * N ])[0] = reinterpret_cast<float4*>(&accum[(i+4) * 8])[0];
        reinterpret_cast<float4*>(&C[offset + (i + 16) * N + 32])[0] = reinterpret_cast<float4*>(&accum[(i+4) * 8 + 4])[0];
    }
}


__global__ void mm_new_3(float* A, float* B, float* C, int N){
    int block_idx = blockIdx.x;
    int block_idy = blockIdx.y;
    int thread_id = threadIdx.x;
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 31;
    int g_row = block_idx * TILE_WIDTH;
    int g_col = block_idy * TILE_WIDTH;

    int sA_row = thread_id >> 1; // 16
    int sA_col = (thread_id & 1) * 4; // 0

    int sB_row = thread_id >> 5; // 1
    int sB_col = (thread_id & 31) * 4; // 0

    int sA_gOffset = sA_row * BLOCK_WIDTH + sA_col;
    int sB_gOffset = sB_row * TILE_WIDTH + sB_col;

    int warp_row = (warp_id / 2) * 32; // 0
    int warp_col = (warp_id % 2) * 64; // 64
    int thread_row = (lane_id / 8) * 4; // 0
    int thread_col = (lane_id % 8) * 4; // 0


    int sA_rOffset = (warp_row + thread_row) * BLOCK_WIDTH; // 0
    int sB_rOffset = warp_col + thread_col; // 64
    int C_gOffset = (warp_row + thread_row) * N + (warp_col + thread_col); // 64

    A = &A[g_row*N];
    B = &B[g_col];
    C = &C[g_row*N + g_col];
    __shared__ float sA[BLOCK_WIDTH * TILE_WIDTH];
    __shared__ float sB[BLOCK_WIDTH * TILE_WIDTH];
    float rA[4];
    float rB[4];

    float fA[8] = {};
    float fB[8] = {};
    float accum[64] = {};
    int j = 0;
    for (int kBlock=0; kBlock<N/BLOCK_WIDTH; kBlock++){
//        sA[sPos] = A[gPos];
//        sB[sPos] = B[gPos];

        //load from gmem
        loadFromGmem_3(A, rA, sA_gOffset);
        loadFromGmem_3(B, rB, sB_gOffset);

        // store to sram
        storeToSmem_3(rA, sA, sA_gOffset);
        storeToSmem_3(rB, sB, sB_gOffset);

        //shift A,B pointers
        __syncthreads();

        if (thread_id==0) {
            //printf("thread is %d, kBlock is %d, kFragment is %d, frag_A is %f\n", 1, kBlock, kFragment, fragment_A[i]);
            for (int i=8;i<16; i++){
                printf("%d %f %f\n", j, sA[i], A[j]);
                j++;
            }
//            for (int i=0;i<8; i++){
//                printf("%d %f\n", j, sA[i]);
//                j++;
//            }
        }

        A += BLOCK_WIDTH;
        B += BLOCK_WIDTH * N;
        // sync thread
        if (thread_id==2) {
            printf("%d\n", sA_gOffset);
        }

//        if (thread_id==0) {
//            //printf("thread is %d, kBlock is %d, kFragment is %d, frag_A is %f\n", 1, kBlock, kFragment, fragment_A[i]);
////            for (int i=8;i<16; i++){
////                printf("%d %f\n", j, sA[i]);
////                j++;
////            }
//            for (int i=0;i<8; i++){
//                printf("%d %f\n", j, sA[i]);
//                j++;
//            }
//        }

        for (int kFragment=0; kFragment<BLOCK_WIDTH; kFragment++) {
//            if (kBlock==0 and block_idx==0 and thread_id==0) {
//
//                printf("kBlock is %d, kFragment is %d, thread id is %d, sB_rOffset is %d, sB[0] is %f\n", kBlock, kFragment, thread_id, sB_rOffset, sB[0]);
//            }
            loadFromSmemA_3(sA, fA, sA_rOffset + kFragment);
            loadFromSmemB_3(sB, fB, sB_rOffset + kFragment * TILE_WIDTH);


            //load from sram
            computeOuterProduct_3(fA, fB, accum);

        }
        __syncthreads();

    }
    storeToGmem_3(accum, C, N, C_gOffset);

}