// naive kernel where each thread computes a single value

#include <hip/hip_runtime.h>
#include <iostream>

#define TILE_WIDTH 16

__global__ void matrix_multiplication(float* A, float* B, float* C, int WIDTH) {

    __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;


    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float s = 0;
    for ( int tile = 0; tile < WIDTH / TILE_WIDTH; tile++){
        Ads[ty][tx] = A[row * WIDTH + tile * TILE_WIDTH + tx];
        Bds[ty][tx] = B[WIDTH * (tile * TILE_WIDTH + ty) + col];
        __syncthreads();

        for ( int k = 0; k < TILE_WIDTH; k++){
            s += Ads[ty][k] * Bds[k][ty];
        }
        __syncthreads();
    }

    if (row < WIDTH && col < WIDTH) {
        C[row * WIDTH + col] = s;
    }

}


int main() {
    int N = 256; // Size of the square matrices
    int M = 16;
    int size = N * N * sizeof(float);

    // Allocate memory on the host
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize matrices h_A and h_B with data
    for (int i=0; i< N*N; i++){
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }
    // Allocate memory on the device
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy matrices h_A and h_B from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define the grid and block dimensions for the kernel launch
    dim3 dimGrid(N/16, N/16); // You can adjust this based on your GPU's capability
    dim3 dimBlock(16, 16);

    // Launch the matrix multiplication kernel
    matrix_multiplication<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    // Copy the result matrix d_C from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i=0; i< N; i++){
        for (int j=0; j< N; j++){
            std::cout << h_C[i*N+j] << " " ;
        }
        std::cout << std::endl;
    }
    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}