// matrix multiplication for 8192 x 8192 x 8192

// global memory coalescing
// shared memory blocking
// register blocking
// vectorized memory load
// global + shared memory pipelining

// https://developer.nvidia.com/blog/cutlass-linear-algebra-cuda/

// thread-tiling: each thread loads 8+8 = 16 floats and computes 8x8 = 64 results
// warp-tiling: each warp computes 64x32 = 2048 results
// block-tiling: each thread block has 2x4 = 8 warps = 256 threads computing 128x128 = 16384 results

// shared memory:
// 128 * 8 * 4 * 2 = 8KB
// registers:
// each thread needs at least 64 * 4 = 256B
// so a threadblock needs at least 256 * 256 = 64 KB

// dim3 dimGrid(16, 16);
// dim3 dimBlock(256, 1);


#include <hip/hip_runtime.h>
#include <iostream>
#define BLOCK_WIDTH 8
#define TILE_WIDTH 128
#define thread_id threadIdx.x
#define warp_id threadIdx.x / 32
#define lane_id threadIdx.x % 32

// warp tiling
#define warp_row (warp_id / 2) * 32
#define warp_col (warp_id % 2) * 64
#define thread_row lane_id / 8
#define thread_col (lane_id % 8) * 4


#define gC_row TILE_WIDTH * blockIdx.y
#define gC_col TILE_WIDTH * blockIdx.x

// shared memory offsets
#define sA_row thread_id / 2
#define sA_col (thread_id % 2) * 4
#define sB_row threadIdx.x / 32
#define sB_col (threadIdx.x % 32) * 4
//
//#define gA_row gC_row + sA_row
//#define gA_col kBlock * BLOCK_WIDTH + sA_col
//#define gB_row kBlock * BLOCK_WIDTH + sB_row
//#define gB_col gC_col + sB_col


__global__ void mm_8(float* A, float* B, float* C, int N){
//    int thread_id = threadIdx.x;
//    int warp_id = threadIdx.x / 32;
//    int lane_id = threadIdx.x % 32;
//
//    int warp_row = (warp_id / 2) * 32;
//    int warp_col = (warp_id % 2) * 64;
//    int thread_row = lane_id / 8;
//    int thread_col = (lane_id % 8) * 4;

    // offset for output matrix C
//    int gC_row =  TILE_WIDTH * blockIdx.y;
//    int gC_col =  TILE_WIDTH * blockIdx.x;

//    int sA_row;
//    int sA_col;
//    int sB_row;
//    int sB_col;


//    int gA_row;
//    int gA_col;
//    int gB_row;
//    int gB_col;

    __shared__ float sA[TILE_WIDTH * BLOCK_WIDTH];
    __shared__ float sB[TILE_WIDTH * BLOCK_WIDTH];

    // fragments
    float fragment_A[8] = {};
    float fragment_B[8] = {};
    float accum[64] = {};

    float buffer_A[8] = {};
    float buffer_B[8] = {};
    int pointer = 0;
    // prologue
    // global -> reg for kBlock = 0
    // reinterpret_cast<float4*>(buffer_A)[0] = reinterpret_cast<float4*>(A)[(gA_row * N + gA_col) / 4];
    // reinterpret_cast<float4*>(buffer_B)[0] = reinterpret_cast<float4*>(B)[(gB_row * N + gB_col) / 4];
    reinterpret_cast<float4*>(buffer_A)[0] = reinterpret_cast<float4*>(A)[((gC_row + sA_row) * N + sA_col) / 4];
    reinterpret_cast<float4*>(buffer_B)[0] = reinterpret_cast<float4*>(B)[(sB_row * N + gC_col + sB_col) / 4];

    // mainloop
    // for kBlock = 0,..., N/BLOCK_WIDTH - 1
    // reg -> shared for kBlock = k
    // global -> reg for kBlock = k + 1
    // FMA for kBlock = k

    for (int kBlock = 0; kBlock < N / BLOCK_WIDTH - 1; kBlock++){
//        gA_row = gC_row + sA_row;
//        gA_col = kBlock * BLOCK_WIDTH + sA_col;
//        gB_row = kBlock * BLOCK_WIDTH + sB_row;
//        gB_col = gC_col + sB_col;




//        reinterpret_cast<float4*>(sA)[(sA_row * BLOCK_WIDTH + sA_col) / 4] = reinterpret_cast<float4*>(A)[(gA_row * N + gA_col) / 4];
//        reinterpret_cast<float4*>(sB)[(sB_row * TILE_WIDTH + sB_col) / 4] = reinterpret_cast<float4*>(B)[(gB_row * N + gB_col) / 4];

        // reg -> shared for kBlock = k
        reinterpret_cast<float4*>(sA)[(sA_row * BLOCK_WIDTH + sA_col) / 4] = reinterpret_cast<float4*>(buffer_A)[pointer];
        reinterpret_cast<float4*>(sB)[(sB_row * TILE_WIDTH + sB_col) / 4] = reinterpret_cast<float4*>(buffer_B)[pointer];

        pointer = (pointer = 0) ? 1 : 0;
        // global -> reg for kBlock = k + 1
        reinterpret_cast<float4*>(buffer_A)[pointer] = reinterpret_cast<float4*>(A)[((gC_row + sA_row) * N + (kBlock + 1) * BLOCK_WIDTH + sA_col) / 4];
        reinterpret_cast<float4*>(buffer_B)[pointer] = reinterpret_cast<float4*>(B)[(((kBlock + 1) * BLOCK_WIDTH + sB_row) * N + gC_col + sB_col) / 4];

        __syncthreads();


        //load a fragment from shared memory to register
        for (int kFragment = 0; kFragment < BLOCK_WIDTH; kFragment++){


            #pragma unroll
            for (int i=0; i<4; i++){
                // Load A fragment, 8 floats
                fragment_A[i] = sA[(warp_row + thread_row + i) * BLOCK_WIDTH + kFragment];
                fragment_A[i+4] = sA[(warp_row + thread_row + 16 + i) * BLOCK_WIDTH + kFragment];

                // Load B fragment, 8 floats
                fragment_B[i] = sB[kFragment * TILE_WIDTH + warp_col + thread_col + i];
                fragment_B[i+4] = sB[kFragment * TILE_WIDTH + warp_col + thread_col + 32 + i];
              }


            // Compute accumulator, 64 floats
            #pragma unroll
            for (int x=0; x<8; x++){
                #pragma unroll
                for (int y=0; y<8; y++){
                    accum[x * 8 + y] += fragment_A[x] * fragment_B[y];
                }
            }

        }
        __syncthreads();
    }


    // epilogue
    // reg -> shared for kBlock = N/BLOCK - 1
    // FMA for kBlock = N/BLOCK - 1

    // reg -> shared for kBlock = N/BLOCK - 1
    reinterpret_cast<float4*>(sA)[(sA_row * BLOCK_WIDTH + sA_col) / 4] = reinterpret_cast<float4*>(buffer_A)[pointer];
    reinterpret_cast<float4*>(sB)[(sB_row * TILE_WIDTH + sB_col) / 4] = reinterpret_cast<float4*>(buffer_B)[pointer];
    // FMA for kBlock = N/BLOCK - 1
    #pragma unroll
    for (int kFragment = 0; kFragment < BLOCK_WIDTH; kFragment++){

        #pragma unroll
        for (int i=0; i<4; i++){
            // Load A fragment, 8 floats
            fragment_A[i] = sA[(warp_row + thread_row + i) * BLOCK_WIDTH + kFragment];
            fragment_A[i+4] = sA[(warp_row + thread_row + 16 + i) * BLOCK_WIDTH + kFragment];

            // Load B fragment, 8 floats
            fragment_B[i] = sB[kFragment * TILE_WIDTH + warp_col + thread_col + i];
            fragment_B[i+4] = sB[kFragment * TILE_WIDTH + warp_col + thread_col + 32 + i];
          }


        // Compute accumulator, 64 floats
        #pragma unroll
        for (int x=0; x<8; x++){
            #pragma unroll
            for (int y=0; y<8; y++){
                accum[x * 8 + y] += fragment_A[x] * fragment_B[y];
            }
        }

    }

    #pragma unroll
    for (int x=0; x<4; x+=1){
        reinterpret_cast<float4*>(C)[((gC_row + warp_row + thread_row + x ) * N + gC_col + warp_col + thread_col) / 4] = reinterpret_cast<float4*>(accum)[(x * 8) /4];
        reinterpret_cast<float4*>(C)[((gC_row + warp_row + thread_row + x ) * N + gC_col + warp_col + thread_col + 32) / 4] = reinterpret_cast<float4*>(accum)[(x * 8 + 4) /4];
        reinterpret_cast<float4*>(C)[((gC_row + warp_row + thread_row + x + 16) * N + gC_col + warp_col + thread_col) / 4] = reinterpret_cast<float4*>(accum)[((x + 4) * 8) /4];
        reinterpret_cast<float4*>(C)[((gC_row + warp_row + thread_row + x + 16) * N + gC_col + warp_col + thread_col + 32) / 4] = reinterpret_cast<float4*>(accum)[((x + 4) * 8 + 4) /4];
    }



}