
#include <hip/hip_runtime.h>
#include <iostream>
#define TILE_WIDTH 128
#define BLOCK_WIDTH 8
#define FLOAT_4(shared_pointer) reinterpret_cast<float4*>(&(shared_pointer))[0]


__global__ void mm_new_6(float* A, float* B, float* C, int N){
    int block_idx = blockIdx.x;
    int block_idy = blockIdx.y;
    int thread_id = threadIdx.x;
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 31;
    int g_row = block_idx * TILE_WIDTH;
    int g_col = block_idy * TILE_WIDTH;

    int sA_row = thread_id >> 1; // 16
    int sA_col = (thread_id & 1) << 2; // 0

    int sB_row = thread_id >> 5; // 1
    int sB_col = (thread_id & 31) << 2; // 0

    int sA_gOffset = sA_row * N + sA_col;
    int sB_gOffset = sB_row * N + sB_col;
    int sA_sOffset = sA_row * BLOCK_WIDTH + sA_col;
    int sB_sOffset = sB_row * TILE_WIDTH + sB_col;

    int warp_row = (warp_id >> 1) << 5; // 0
    int warp_col = (warp_id & 1) << 6; // 64
    int thread_row = (lane_id >> 3) << 2; // 0
    int thread_col = (lane_id & 7) << 2; // 0


    int sA_rOffset = (warp_row + thread_row) * BLOCK_WIDTH; // 0
    int sB_rOffset = warp_col + thread_col; // 64
    int C_gOffset = (warp_row + thread_row) * N + (warp_col + thread_col); // 64

    A = &A[g_row*N];
    B = &B[g_col];
    C = &C[g_row*N + g_col];

    __shared__ float sA[2][BLOCK_WIDTH * TILE_WIDTH];
    __shared__ float sB[2][BLOCK_WIDTH * TILE_WIDTH];

    float rA[2][4];
    float rB[2][4];

    float fA[2][8];
    float fB[2][8];
    float accum[8][8];

    for(int i=0; i<8; i++){

        for(int j=0; j<8; j++){
            accum[i][j]=0.0;
        }
    }

    int shared_pointer = 0;
    int reg_pointer = 0;
    // prologue, load kBLock = 0 from global to shared
    //load from gmem A, B
    FLOAT_4(rA[shared_pointer][0]) = FLOAT_4(A[sA_gOffset]);
    FLOAT_4(rB[shared_pointer][0]) = FLOAT_4(B[sB_gOffset]);

    // store to smem sA, sB
    FLOAT_4(sA[shared_pointer][sA_sOffset]) = FLOAT_4(rA[shared_pointer][0]);
    FLOAT_4(sB[shared_pointer][sB_sOffset]) = FLOAT_4(rB[shared_pointer][0]);


    //shift A,B shared_pointers
    A += BLOCK_WIDTH;
    B += BLOCK_WIDTH * N;

    __syncthreads();
//    if (block_idx==0 and block_idy==0 and thread_id ==0) {
//        printf("%f %f %f", rA[shared_pointer][0], sA[shared_pointer][0], sB[shared_pointer][0]);
//    }

    //mainloop
    // compute kblock = 0,..., N/BLOCK_WIDTH - 2
    // load kblock = 1,..., N/BLOCK_WIDTH - 1
    for (int kBlock=0; kBlock<N/BLOCK_WIDTH; kBlock++){

        //load from gmem for next block
        if (kBlock < N/BLOCK_WIDTH - 1) {

            //load from gmem A, B
            FLOAT_4(rA[shared_pointer ^ 1][0]) = FLOAT_4(A[sA_gOffset]);
            FLOAT_4(rB[shared_pointer ^ 1][0]) = FLOAT_4(B[sB_gOffset]);
            
            // store to smem sA, sB
            FLOAT_4(sA[shared_pointer ^ 1][sA_sOffset]) = FLOAT_4(rA[shared_pointer ^ 1][0]);
            FLOAT_4(sB[shared_pointer ^ 1][sA_sOffset]) = FLOAT_4(rB[shared_pointer ^ 1][0]);

        }


        //
        #pragma unroll
        for (int i=0; i<4; i++) {
                fA[reg_pointer][i] = sA[shared_pointer][sA_rOffset + i * BLOCK_WIDTH];
                fA[reg_pointer][i+4] = sA[shared_pointer][sA_rOffset + (i + 16) * BLOCK_WIDTH];
                fB[reg_pointer][i] = sB[shared_pointer][sB_rOffset + i];
                fB[reg_pointer][i+4] = sB[shared_pointer][sB_rOffset + i + 32];
        }


        for (int kFragment=0; kFragment<BLOCK_WIDTH; kFragment++) {


            if (kFragment < BLOCK_WIDTH -1) {
                // load from smem A,B for next tile
                for (int i=0; i<4; i++) {
                    fA[reg_pointer ^ 1][i] = sA[shared_pointer][sA_rOffset + kFragment + 1 + i * BLOCK_WIDTH];
                    fA[reg_pointer ^ 1][i+4] = sA[shared_pointer][sA_rOffset + kFragment + 1 + (i + 16) * BLOCK_WIDTH];
                    fB[reg_pointer ^ 1][i] = sB[shared_pointer][sB_rOffset + (kFragment + 1) * TILE_WIDTH + i];
                    fB[reg_pointer ^ 1][i+4] = sB[shared_pointer][sB_rOffset + (kFragment + 1) * TILE_WIDTH + i + 32];
                }
            }


            // compute outer product

            for (int i = 0; i < 8; i++) {
                for (int j = 0; j < 8; j++) { 
                    accum[i][j] += fA[reg_pointer][i] * fB[reg_pointer][j];
                }   
            }

            reg_pointer ^= 1;

        }

        // store to smem for next block
        if (kBlock < N/BLOCK_WIDTH - 1) {

            // store to smem sA, sB
            FLOAT_4(sA[shared_pointer ^ 1][sA_sOffset]) = FLOAT_4(rA[shared_pointer ^ 1][0]);
            FLOAT_4(sB[shared_pointer ^ 1][sA_sOffset]) = FLOAT_4(rB[shared_pointer ^ 1][0]);

            A += BLOCK_WIDTH;
            B += BLOCK_WIDTH * N;
        }

        shared_pointer ^= 1;
        
        __syncthreads();

    }


    // store to gmem C

    for (int i=0;i<4;i++) {
        FLOAT_4(C[C_gOffset + i * N]) = FLOAT_4(accum[i][0]);
        FLOAT_4(C[C_gOffset + i * N + 32]) = FLOAT_4(accum[i][4]);
        FLOAT_4(C[C_gOffset + (i + 16) * N ]) = FLOAT_4(accum[i+4][0]);
        FLOAT_4(C[C_gOffset + (i + 16) * N + 32]) = FLOAT_4(accum[i+4][4]);
    }



}