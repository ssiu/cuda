//https://github.com/siboehm/SGEMM_CUDA/blob/master/cuBLAS_sgemm.cu
#include <iostream>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>



void mm_cublas(thrust::device_vector<float> A, thrust::device_vector<float> B, thrust::device_vector<float> C, int N) {
    hipError_t cudaStat;  // hipMalloc status
    hipblasStatus_t stat;   // cuBLAS functions status
    hipblasHandle_t handle; // cuBLAS context

    float alpha = 1.0f;
    float beta = 1.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N,
                     d_B, N, &beta, d_C, N);

}