//https://github.com/siboehm/SGEMM_CUDA/blob/master/cuBLAS_sgemm.cu
#include <iostream>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>



void mm_cublas(thrust::device_vector<float> dA, thrust::device_vector<float> dB, thrust::device_vector<float> dC, int N) {
    hipError_t cudaStat;  // hipMalloc status
    hipblasStatus_t stat;   // cuBLAS functions status
    hipblasHandle_t handle; // cuBLAS context

    float alpha = 1.0f;
    float beta = 1.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, dA, N,
                     dB, N, &beta, dC, N);

}