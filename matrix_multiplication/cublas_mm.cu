//https://github.com/siboehm/SGEMM_CUDA/blob/master/cuBLAS_sgemm.cu

#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

/*
 * A stand-alone script to invoke & benchmark standard cuBLAS SGEMM performance
 */

int main(int argc, char *argv[]) {

    int N = 2048; // Size of the square matrices
    int size = N * N * sizeof(float);
    hipError_t cudaStat;  // hipMalloc status
    hipblasStatus_t stat;   // cuBLAS functions status
    hipblasHandle_t handle; // cuBLAS context


    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize matrices h_A and h_B with data
    for (int i=0; i< N*N; i++){
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
        h_C[i] = 0.0f;
    }

    // Allocate memory on the device
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    stat = hipblasCreate(&handle); // initialize CUBLAS context

    // Copy matrices h_A and h_B from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    float alpha = 1.0f;
    float beta = 1.0f;

    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N,
                     d_B, N, &beta, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i=0; i< N; i++){
        for (int j=0; j< N; j++){
            std::cout << h_C[i*N+j] << " " ;
        }
        std::cout << std::endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle); // destroy CUBLAS context
    free(h_A);
    free(h_B);
    free(h_C);

    return EXIT_SUCCESS;
}