#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <random>
#include "mm.cuh"
#include "utils.cuh"


int main(){
    int N = 2048;

    thrust::host_vector<float> hA = generateMatrices(N);
    thrust::host_vector<float> hB = generateMatrices(N);
    thrust::host_vector<float> hC(N*N);
    thrust::host_vector<float> hC_cublas(N*N);

    thrust::device_vector<float> dA = hA;
    thrust::device_vector<float> dB = hB;
    thrust::device_vector<float> dC = hC;
    thrust::device_vector<float> dC_cublas(N*N);


    dim3 dimGrid(64, 64);
    dim3 dimBlock(32, 32);
//    mm_0<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
//                                    thrust::raw_pointer_cast(dC.data()), N);
//
    mm_1<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
                                    thrust::raw_pointer_cast(dC.data()), N);
    mm_2<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
                                    thrust::raw_pointer_cast(dC.data()), N);
//
//
    dim3 dimGrid3(64, 64);
    dim3 dimBlock3(8, 32);
    mm_3<<<dimGrid3, dimBlock3>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
                                    thrust::raw_pointer_cast(dC.data()), N);

    dim3 dimGrid4(16, 16);
    dim3 dimBlock4(256, 1);
    mm_4<<<dimGrid4, dimBlock4>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
                                   thrust::raw_pointer_cast(dC.data()), N);

//    mm_5<<<dimGrid4, dimBlock4>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
//                                   thrust::raw_pointer_cast(dC.data()), N);
//
////// Device code
////__global__ void MyKernel(...)
////{
////    extern __shared__ float buffer[];
////    ...
////}
////
////// Host code
//    int maxbytes = 98304; // 96 KB
//    hipFuncSetAttribute(reinterpret_cast<const void*>(mm_6), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
//    mm_6<<<dimGrid4, dimBlock4, maxbytes>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
//                                   thrust::raw_pointer_cast(dC.data()), N);
    mm_7<<<dimGrid4, dimBlock4>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
                               thrust::raw_pointer_cast(dC.data()), N);

    hC = dC;


    //
    // cublas
    //
    float alpha = 1.0f;
    float beta = 1.0f;

    hipError_t cudaStat;  // hipMalloc status
    hipblasStatus_t stat;   // cuBLAS functions status
    hipblasHandle_t handle; // cuBLAS context
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, thrust::raw_pointer_cast(dB.data()), N,
                     thrust::raw_pointer_cast(dA.data()), N, &beta, thrust::raw_pointer_cast(dC_cublas.data()), N);

    hC_cublas = dC_cublas;

    hipblasDestroy(handle);
    //
    //
    //




    if (isSameMatrices(hC.data(), hC_cublas.data(), N)==0){
//        for (int i=0;i<N;i += 128){
//            for (int j=0;j<N; j+=128){
//                std::cout << N * i + j << " " << hC[N * i + j] << " " << hC_cublas[N * i + j] << std::endl;
//            }
//        }

        for (int i=0;i<100;i++){
            std::cout << i << " " << hC[i] << " " << hC_cublas[i] << std::endl;
        }
//        int num = countZeros(hC.data(), N);
//        std::cout << "number of zeros in hC is " << num << std::endl;
        std::cout << "Wrong answer" << std::endl;
    } else {
        std::cout << "Correct answer" << std::endl;
    }

    return 0;
}