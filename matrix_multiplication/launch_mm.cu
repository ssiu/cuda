#include <iostream>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <random>
#include "mm.cuh"
#include "utils.cuh"


int main(){
    int N = 2048;

    thrust::host_vector<float> hA = generateMatrices(N);
    thrust::host_vector<float> hB = generateMatrices(N);
    thrust::host_vector<float> hC(N*N);
    thrust::host_vector<float> hC_cublas(N*N);

    thrust::device_vector<float> dA = hA;
    thrust::device_vector<float> dB = hB;
    thrust::device_vector<float> dC = hC;
    thrust::device_vector<float> dC_cublas(N*N);


//    dim3 dimGrid(32, 32);
//    dim3 dimBlock(32, 32);
//    mm_0<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
//                                    thrust::raw_pointer_cast(dC.data()), N);
//
//    mm_1<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
//                                    thrust::raw_pointer_cast(dC.data()), N);
//    mm_2<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
//                                    thrust::raw_pointer_cast(dC.data()), N);
//
//
//    dim3 dimGrid3(64, 64);
//    dim3 dimBlock3(8, 32);
//    mm_3<<<dimGrid3, dimBlock3>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
//                                    thrust::raw_pointer_cast(dC.data()), N);




    //
    // cublas
    //
    float alpha = 1.0f;
    float beta = 1.0f;

    hipError_t cudaStat;  // hipMalloc status
    hipblasStatus_t stat;   // cuBLAS functions status
    hipblasHandle_t handle; // cuBLAS context
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, thrust::raw_pointer_cast(dB.data()), N,
                     thrust::raw_pointer_cast(dA.data()), N, &beta, thrust::raw_pointer_cast(dC_cublas.data()), N);

    hC_cublas = dC_cublas;

    hipblasDestroy(handle);
    //
    //
    //

    dim3 dimGrid4(16, 16);
    dim3 dimBlock4(256, 1);
    mm_4<<<dimGrid4, dimBlock4>>>(thrust::raw_pointer_cast(dA.data()), thrust::raw_pointer_cast(dB.data()),
                                   thrust::raw_pointer_cast(dC.data()), N);

    hC = dC;


    if (isSameMatrices(hC.data(), hC_cublas.data(), N)==0){
        for (int i=0;i<2048;i += 128){
            for (int j=0;j<2048; j+=128){
                std::cout << 2048 * i + j << " " << hC[2048 * i + j] << " " << hC_cublas[2048 * i + j] << std::endl;
            }
        }

//        for (int i=0;i<100;i++){
//            std::cout << hC[i] << " " << hC_cublas[i] << std::endl;
//        }
        num = countZeros(hC);
        std::cout << "number of zeros in hC is "<< num << std::endl;
        std::cout << "Wrong answer" << std::endl;
    } else {
        std::cout << "Correct answer" << std::endl;
    }

    return 0;
}