#include <iostream>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "mm.h"


thrust::host_vector<float> generateMatrices(int N) {
    thrust::host_vector<float> A(N * N);

    // Create random engine
    std::random_device rd;
    std::mt19937 gen(rd());

    // Define distribution range
    std::uniform_real_distribution<float> dis(0.0, 1.0);

    // Generate random matrix
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            float randomFloat = dis(gen);
            A[i * N + j] = randomFloat;
        }
    }

    // Return both matrices
    return A;
}


int main(){
    int N = 2048;

    thrust::host_vector<float> hA = generateMatrices(N);
    thrust::host_vector<float> hB = generateMatrices(N);
    thrust::host_vector<float> hC;

    thrust::device_vector<float> dA = hA;
    thrust::device_vector<float> dB = hB;
    thrust::device_vector<float> dC = hC;


    mm_cublas(dA, dB, dC, N);

}