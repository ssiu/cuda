
#include <hip/hip_runtime.h>
// naive matrix multiplication

__global__ void mm_0(float* A, float* B, float*C, int N){
    //int threadId = threadIdx.x + blockDim.x * threadIdx.y
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;

    for (int i = 0; i< N; i++){
        C[row*N + col] = A[row*N + i] * B[i*N + col];
    }
}