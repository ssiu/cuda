// naive matrix multiplication

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mm_0(float* A, float* B, float* C, int N){
    //int threadId = threadIdx.x + blockDim.x * threadIdx.y
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    int sum = 0;
    for (int i = 0; i < N; i++){
        sum += A[row*N + i] * B[i*N + col];
    }
    C[row*N + col] = sum;
}