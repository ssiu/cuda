// naive matrix multiplication

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mm_0(float* A, float* B, float* C, int N){
    //int threadId = threadIdx.x + blockDim.x * threadIdx.y
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    int sum = 0;
    for (int i = 0; i < N; i++){
        if (blockIdx.x == 0 and blockIdx.x==0 and threadIdx.x==0 and threadIdx.y==0){
            printf("A IS %f, B is %f\n", A[row*N + i], B[i*N + col]);
        }
        sum += A[row*N + i] * B[i*N + col];
    }
    C[row*N + col] = sum;
}