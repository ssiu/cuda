
#include <hip/hip_runtime.h>
#include <iostream>
#define A(i,j) A[(i) * N + (j)]
#define B(i,j) B[(i) * N + (j)]
#define C(i,j) C[(i) * N + (j)]
#define sA(pointer, i,j) sA[(pointer)][((i) << 7) + (j)]
#define sB(pointer, i,j) sB[(pointer)][((i) << 7) + (j)]
#define TILE_WIDTH 128
#define BLOCK_WIDTH 8
#define FLOAT_4(pointer) reinterpret_cast<float4*>(&(pointer))[0]


__global__ __launch_bounds__(256)
void mm_new_9(float* A, float* B, float* C, int N){
    int thread_id = threadIdx.x;
    int block_idx = blockIdx.x;
    int block_idy = blockIdx.y;
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 31;
    int warp_row = (warp_id >> 1) << 5;

    int warp_col = (warp_id & 1) << 6;
    int thread_row = (lane_id >> 3) << 2;

    int thread_col = (lane_id & 7) << 2;

    int sA_row = thread_id >> 1;
    int sA_col = (thread_id & 1) << 2;

    int sB_row = thread_id >> 5;
    int sB_col = (thread_id & 31) << 2;

    int C_row = warp_row + thread_row;
    int C_col = warp_col + thread_col;


//    int sA_rOffset = warp_row + thread_row; // 0
//    int sB_rOffset = warp_col + thread_col; // 64
//    int C_gOffset = (warp_row + thread_row) * N + (warp_col + thread_col); // 64

    A = &A((block_idx << 7), 0);
    B = &B(0, (block_idy << 7));
    C = &C((block_idx << 7), (block_idy << 7));

    __shared__ float sA[2][BLOCK_WIDTH * TILE_WIDTH];
    __shared__ float sB[2][BLOCK_WIDTH * TILE_WIDTH];


    float rA[4];
    float rB[4];

    float fA[2][8] = {};
    float fB[2][8] = {};

    float accum[64] = {};

    int shared_pointer = 0;
    int reg_pointer = 0;

    // load first block
    FLOAT_4(rA) = FLOAT_4(A(sA_row, sA_col));
    FLOAT_4(rB) = FLOAT_4(B(sB_row, sB_col));
    #pragma unroll
    for (int i=0; i<4;i++){
        sA(shared_pointer, sA_col + i, sA_row) = rA[i];
        // sA[shared_pointer][sA_sOffset + i*TILE_WIDTH] = rA[i];
    }

    FLOAT_4(sB(shared_pointer, sB_row, sB_col)) = FLOAT_4(rB);

    __syncthreads();

    A += BLOCK_WIDTH;
    B += BLOCK_WIDTH * N;


    // load second block
//    FLOAT_4(rA) = FLOAT_4(A(sA_row, sA_col));
//    FLOAT_4(rB) = FLOAT_4(B(sB_row, sB_col));

    for (int kBlock=0; kBlock<N/BLOCK_WIDTH; kBlock++){

        // load from gmem A, B for next block
        if (kBlock < N/BLOCK_WIDTH - 1) {
            FLOAT_4(rA) = FLOAT_4(A(sA_row, sA_col));
            FLOAT_4(rB) = FLOAT_4(B(sB_row, sB_col));
        }


        // load from smem A, B
        FLOAT_4(fA[reg_pointer][0]) = FLOAT_4(sA(shared_pointer, 0, C_row));
        FLOAT_4(fA[reg_pointer][4]) = FLOAT_4(sA(shared_pointer, 0, C_row + 16));
        FLOAT_4(fB[reg_pointer][0]) = FLOAT_4(sB(shared_pointer, 0, C_col));
        FLOAT_4(fB[reg_pointer][4]) = FLOAT_4(sB(shared_pointer, 0, C_col + 32));

        #pragma unroll
        for (int kFragment=0; kFragment<BLOCK_WIDTH; kFragment++) {

            if (kFragment < BLOCK_WIDTH - 1 ) {
                // load next fragment from smem A, B
                FLOAT_4(fA[reg_pointer^1][0]) = FLOAT_4(sA(shared_pointer, kFragment+1, C_row));
                FLOAT_4(fA[reg_pointer^1][4]) = FLOAT_4(sA(shared_pointer, kFragment+1, C_row + 16));
                FLOAT_4(fB[reg_pointer^1][0]) = FLOAT_4(sB(shared_pointer, kFragment+1, C_col));
                FLOAT_4(fB[reg_pointer^1][4]) = FLOAT_4(sB(shared_pointer, kFragment+1, C_col + 32));
            }


            // compute outer product
            #pragma unroll
            for (int i=0; i<8;i++){
                #pragma unroll
                for (int j=0; j<8; j++) {
                    accum[i*8+j] += fA[reg_pointer][i] * fB[reg_pointer][j];
                }
             }
            reg_pointer ^= 1;
        }





        // store to smem sA, sB for next block
        if (kBlock < N/BLOCK_WIDTH - 1) {

            //FLOAT_4(sA[sA_sOffset]) = FLOAT_4(rA);
            #pragma unroll
            for (int i=0; i<4;i++){
                sA(shared_pointer^1, sA_col + i, sA_row) = rA[i];
                //sA[shared_pointer^1][sA_sOffset + i*TILE_WIDTH] = rA[i];
            }

            FLOAT_4(sB(shared_pointer^1, sB_row, sB_col)) = FLOAT_4(rB);

            __syncthreads();

            A += BLOCK_WIDTH;
            B += BLOCK_WIDTH * N;

            shared_pointer ^= 1;
        }

    }

//    storeToGmem_5(accum, C, N, C_gOffset);

    // store to gmem C
    #pragma unroll
    for (int i=0;i<4;i++) {

        FLOAT_4(C(C_row + i, C_col)) = FLOAT_4(accum[i * 8]);
        FLOAT_4(C(C_row + i, C_col + 32)) = FLOAT_4(accum[i * 8 + 4]);
        FLOAT_4(C(C_row + i + 16, C_col)) = FLOAT_4(accum[(i+4) * 8]);
        FLOAT_4(C(C_row + i + 16, C_col + 32)) = FLOAT_4(accum[(i+4) * 8 + 4]);

    }



}