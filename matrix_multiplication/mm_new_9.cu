
#include <hip/hip_runtime.h>
#include <iostream>
#define TILE_WIDTH 128
#define BLOCK_WIDTH 8
#define FLOAT_4(pointer) reinterpret_cast<float4*>(&(pointer))[0]


__global__ void mm_new_9(float* A, float* B, float* C, int N){
    int block_idx = blockIdx.x;
    int block_idy = blockIdx.y;
    int thread_id = threadIdx.x;
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 31;
    int g_row = block_idx * TILE_WIDTH;
    int g_col = block_idy * TILE_WIDTH;

    int sA_row = thread_id >> 1; // 16
    int sA_col = (thread_id & 1) << 2; // 0

    int sB_row = thread_id >> 5; // 1
    int sB_col = (thread_id & 31) << 2; // 0

    int sA_gOffset = sA_row * N + sA_col;
    int sB_gOffset = sB_row * N + sB_col;
    // need to transpose A tile
    //int sA_sOffset = sA_row * BLOCK_WIDTH + sA_col;
    int sA_sOffset = sA_col * TILE_WIDTH + sA_row;
    int sB_sOffset = sB_row * TILE_WIDTH + sB_col;

    int warp_row = (warp_id >> 1) << 5; // 0
    int warp_col = (warp_id & 1) << 6; // 64
    int thread_row = (lane_id >> 3) << 2; // 0
    int thread_col = (lane_id & 7) << 2; // 0


    int sA_rOffset = warp_row + thread_row; // 0
    int sB_rOffset = warp_col + thread_col; // 64
    int C_gOffset = (warp_row + thread_row) * N + (warp_col + thread_col); // 64

    A = &A[g_row*N];
    B = &B[g_col];
    C = &C[g_row*N + g_col];

    __shared__ float sA[2][BLOCK_WIDTH * TILE_WIDTH];
    __shared__ float sB[2][BLOCK_WIDTH * TILE_WIDTH];


    float rA[4];
    float rB[4];

    float fA[2][8] = {};
    float fB[2][8] = {};
    float accum[64] = {};

    int shared_pointer = 0;
    int reg_pointer = 0;
    // load first block
    FLOAT_4(rA) = FLOAT_4(A[sA_gOffset]);
    FLOAT_4(rB) = FLOAT_4(B[sB_gOffset]);

    for (int i=0; i<4;i++){
        sA[shared_pointer][sA_sOffset + i*TILE_WIDTH] = rA[i];
    }

    FLOAT_4(sB[shared_pointer][sB_sOffset]) = FLOAT_4(rB);

    __syncthreads();

    A += BLOCK_WIDTH;
    B += BLOCK_WIDTH * N;


    for (int kBlock=0; kBlock<N/BLOCK_WIDTH; kBlock++){
//        sA[sPos] = A[gPos];
//        sB[sPos] = B[gPos];

        // load from gmem A, B for next block
        if (kBlock < N/BLOCK_WIDTH - 1) {

            FLOAT_4(rA) = FLOAT_4(A[sA_gOffset]);
            FLOAT_4(rB) = FLOAT_4(B[sB_gOffset]);
        }

        // load first fragment
        FLOAT_4(fA[reg_pointer][0]) = FLOAT_4(sA[shared_pointer][sA_rOffset]);
        FLOAT_4(fA[reg_pointer][4]) = FLOAT_4(sA[shared_pointer][sA_rOffset+ 16]);
        FLOAT_4(fB[reg_pointer][0]) = FLOAT_4(sB[shared_pointer][sB_rOffset]);
        FLOAT_4(fB[reg_pointer][4]) = FLOAT_4(sB[shared_pointer][sB_rOffset + 32]);

        for (int kFragment=0; kFragment<BLOCK_WIDTH; kFragment++) {

            // load next fragment
            if (kFragment < BLOCK_WIDTH - 1) {
                FLOAT_4(fA[reg_pointer ^ 1][0]) = FLOAT_4(sA[shared_pointer][sA_rOffset + (kFragment + 1) * TILE_WIDTH]);
                FLOAT_4(fA[reg_pointer ^ 1][4]) = FLOAT_4(sA[shared_pointer][sA_rOffset + (kFragment + 1) * TILE_WIDTH + 16]);
                FLOAT_4(fB[reg_pointer ^ 1][0]) = FLOAT_4(sB[shared_pointer][sB_rOffset + (kFragment + 1) * TILE_WIDTH]);
                FLOAT_4(fB[reg_pointer ^ 1][4]) = FLOAT_4(sB[shared_pointer][sB_rOffset + (kFragment + 1) * TILE_WIDTH + 32]);
            }


            // compute outer product
            for (int i=0; i<8;i++){
                for (int j=0; j<8; j++) {
                    accum[i*8+j] += fA[reg_pointer][i] * fB[reg_pointer][j];
                }
            }

            reg_pointer ^= 1;
        }

        // store to smem sA, sB for next block
        if (kBlock < N/BLOCK_WIDTH - 1) {


            //FLOAT_4(sA[sA_sOffset]) = FLOAT_4(rA);
            for (int i=0; i<4;i++){
                sA[shared_pointer^1][sA_sOffset + i*TILE_WIDTH] = rA[i];
            }

            FLOAT_4(sB[shared_pointer^1][sB_sOffset]) = FLOAT_4(rB);

            __syncthreads();

            A += BLOCK_WIDTH;
            B += BLOCK_WIDTH * N;

            shared_pointer ^= 1;
        }

    }

//    storeToGmem_5(accum, C, N, C_gOffset);

    // store to gmem C
    for (int i=0;i<4;i++) {
        FLOAT_4(C[C_gOffset + i * N]) = FLOAT_4(accum[i * 8]);
        FLOAT_4(C[C_gOffset + i * N + 32]) = FLOAT_4(accum[i * 8 + 4]);
        FLOAT_4(C[C_gOffset + (i + 16) * N ]) = FLOAT_4(accum[(i+4) * 8]);
        FLOAT_4(C[C_gOffset + (i + 16) * N + 32]) = FLOAT_4(accum[(i+4) * 8 + 4]);
    }



}